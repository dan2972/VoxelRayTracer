#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <chrono>
#include "shader.h"
#include "resource_manager.h"
#include "model.h"
#include "camera.cuh"
#include "raytracer.cuh"
#include "bbox.cuh"
#include "world.cuh"

void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods) {
    if (key == GLFW_KEY_ESCAPE && action == GLFW_PRESS) {
        // glfwSetWindowShouldClose(window, GLFW_TRUE);
        if (glfwGetInputMode(window, GLFW_CURSOR) == GLFW_CURSOR_NORMAL)
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
        else
            glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_NORMAL);
    }
}

int main() {
    const int width = 1920;
    const int height = 1080;

    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return -1;
    }

    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(width, height, "CUDA + OpenGL Example", nullptr, nullptr);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return -1;
    }

    glfwMakeContextCurrent(window);

    glfwSetKeyCallback(window, keyCallback);
    glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
    // glfwSwapInterval(0); // Disable VSync

    // Initialize Glad
    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
        std::cerr << "Failed to initialize Glad" << std::endl;
        return -1;
    }

    // OpenGL Quad Data
    std::vector<float> quadVertices = {
        // positions    // texCoords
        -1.0f, -1.0f,  0.0f, 0.0f,
         1.0f, -1.0f,  1.0f, 0.0f,
         1.0f,  1.0f,  1.0f, 1.0f,
        -1.0f,  1.0f,  0.0f, 1.0f
    };

    std::vector<unsigned int> quadIndices = {
        0, 1, 2,
        0, 2, 3
    };

    Model screenQuad;
    screenQuad.addData({quadVertices, quadIndices}, {2, 2});

    // Compile Shaders and Link Program
    Shader screen_quad_shader = ResourceManager::loadShader("res/shaders/basic_quad.vert", "res/shaders/basic_quad.frag", nullptr, "screen_quad");

    Texture2D screenQuadTexture;
    screenQuadTexture.generate(width, height, nullptr);
    ResourceManager::addTexture(screenQuadTexture, "screenQuadTexture");

    // Register OpenGL Texture with CUDA
    hipGraphicsResource* cudaResource;
    checkCudaErrors(hipGraphicsGLRegisterImage(&cudaResource, screenQuadTexture.ID, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

    // Allocate CUDA memory once
    uchar4* devPixels;
    checkCudaErrors(hipMalloc(&devPixels, width * height * sizeof(uchar4)));

    // FPS Counter Variables
    auto lastTime = std::chrono::high_resolution_clock::now();
    int frames = 0;
    
    Camera** camera;
    hiprandState *dRandState;
    World** dWorld;
    checkCudaErrors(hipMalloc((void **)&camera, sizeof(Camera *)));
    checkCudaErrors(hipMalloc((void **)&dRandState, width*height*sizeof(hiprandState)));
    checkCudaErrors(hipMalloc((void **)&dWorld, sizeof(World *)));

    create_world<<<1, 1>>>(dWorld, camera, width, height);
    checkCudaErrors(hipGetLastError());

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    renderInit<<<gridSize, blockSize>>>(width, height, dRandState);
    checkCudaErrors(hipGetLastError());

    std::cout << "Starting main loop" << std::endl;

    double lastxpos, lastypos;
    glfwGetCursorPos(window, &lastxpos, &lastypos);
    int prevMouseClickState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);

    // Main Loop
    while (!glfwWindowShouldClose(window)) {

        Vec3 cameraDeltaPos(0,0,0);
        Vec3 cameraDeltaRotation(0,0,0);
        if (glfwGetKey(window, GLFW_KEY_D) == GLFW_PRESS) {
            cameraDeltaPos[0] += 1;
        }
        if (glfwGetKey(window, GLFW_KEY_A) == GLFW_PRESS) {
            cameraDeltaPos[0] -= 1;
        }
        if (glfwGetKey(window, GLFW_KEY_SPACE) == GLFW_PRESS) {
            cameraDeltaPos[1] += 1;
        }
        if (glfwGetKey(window, GLFW_KEY_LEFT_SHIFT) == GLFW_PRESS) {
            cameraDeltaPos[1] -= 1;
        }
        if (glfwGetKey(window, GLFW_KEY_W) == GLFW_PRESS) {
            cameraDeltaPos[2] += 1;
        }
        if (glfwGetKey(window, GLFW_KEY_S) == GLFW_PRESS) {
            cameraDeltaPos[2] -= 1;
        }

        double xpos, ypos;
        glfwGetCursorPos(window, &xpos, &ypos);
        float xoffset = xpos - lastxpos;
        float yoffset = lastypos - ypos;
        lastxpos = xpos;
        lastypos = ypos;

        int mouseClickState = glfwGetMouseButton(window, GLFW_MOUSE_BUTTON_LEFT);
        if (mouseClickState == GLFW_PRESS && prevMouseClickState == GLFW_RELEASE) {
            placeBlock<<<1, 1>>>(camera, dWorld, 20);
        }
        prevMouseClickState = mouseClickState;

        controlCamera<<<1, 1>>>(camera, xoffset, yoffset, cameraDeltaPos);
        
        // Map CUDA Resource
        hipArray* textureArray;
        checkCudaErrors(hipGraphicsMapResources(1, &cudaResource, 0));
        checkCudaErrors(hipGraphicsSubResourceGetMappedArray(&textureArray, cudaResource, 0, 0));
        
        render<<<gridSize, blockSize>>>(devPixels, width, height, camera, dWorld, dRandState);
        checkCudaErrors(hipMemcpyToArray(textureArray, 0, 0, devPixels, width * height * sizeof(uchar4), hipMemcpyDeviceToDevice));

        checkCudaErrors(hipGraphicsUnmapResources(1, &cudaResource, 0));

        // Render
        glClear(GL_COLOR_BUFFER_BIT);
        screen_quad_shader.use();
        screenQuad.bind();
        screenQuadTexture.bind();
        glDrawElements(GL_TRIANGLES, screenQuad.getRenderInfo().indicesCount, GL_UNSIGNED_INT, 0);

        glfwSwapBuffers(window);
        glfwPollEvents();

        // Update FPS Counter
        frames++;
        auto currentTime = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed = currentTime - lastTime;
        if (elapsed.count() >= 1.0) {
            std::cout << "FPS: " << frames << std::endl;
            frames = 0;
            lastTime = currentTime;
        }
    }

    // Cleanup
    hipGraphicsUnregisterResource(cudaResource);
    checkCudaErrors(hipFree(devPixels));

    glfwDestroyWindow(window);
    glfwTerminate();

    return 0;
}
